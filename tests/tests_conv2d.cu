#include "hip/hip_runtime.h"
#include <stdio.h>
#include "basics/tensor.cu"
#include "initializers/gaussian_kernel_initializer.cu"
#include <assert.h>
#include <cmath>
#include <vector>
#include "basics/session.hpp"
#include "layers/conv2d.cu"
#include "tmp/bitmap_image.hpp"


void test_conv2d_cpu() {
  printf("Example code for conv2d cpu\n");
  size_t h = 400;
  size_t w = 400;

  Session* session = Session::GetNewSession();
  session->gpu = false;
 
  // inputs: filter_height, filter_width, in_channels, out_channels, stride
  Conv2D<float> * conv_layer = new Conv2D<float>(15,15,1,1,2,new GaussianKernelInitializer<float>(15));
  const char* OUTPUT_BMP_PATH = "./tmp/test/out.bmp";

  size_t b_dims[4] = {1, h, w, 1};
  Tensor<float>* bottom = Tensor<float>::CreateTensorCPU(b_dims);
  size_t t_dims[4] = {1, h/2, w/2, 1};
  Tensor<float>* top = Tensor<float>::CreateTensorCPU(t_dims);

  for(int i = 0; i < h; i++) {
  	for(int j = 0; j < w; j++) {
  	  int b_idx[4] = {0, i, j, 0};
  	  bottom->at(b_idx) = (float) ((i+j) % 255);
  	}
  }
  conv_layer->Forward({bottom}, {top});

  bitmap_image img(w/2, h/2);
  for (int i = 0; i < h/2; i++) {
    for (int j = 0; j < w/2; j++) {
      unsigned val = (unsigned) top->at(0, i, j, 0);
      img.set_pixel(j, i, val, val, val);
    }
  }
  img.save_image(OUTPUT_BMP_PATH);
  delete conv_layer;
}


__global__ void init_bottom(Tensor<float> * bottom) {
  for(int i = 0; i < bottom->GetDims()[1]; i++) {
  	for(int j = 0; j < bottom->GetDims()[2]; j++) {
  	  int b_idx[4] = {0, i, j, 0};
  	  bottom->at(b_idx) = (float) ((i+j) % 255);
  	}
  }
}

__global__ void show_top(Tensor<float>* top) {
  size_t h = top->GetDims()[1];
  size_t w = top->GetDims()[2];
  for (int i = 0; i < h; i++) {
    for (int j = 0; j < w; j++) {
  	  printf("%f ", top->at(0, i, j, 0));
    }
    printf("\n");
  }	
  printf("%d \n", top->GetDataPtr());

}

void test_conv2d_gpu() {
  printf("Example code for conv2d gpu\n");
  hipError_t cudaStatus = hipSetDevice(0);
  checkCudaErrors(cudaStatus);

  size_t h = 400;
  size_t w = 400;

  Session* session = Session::GetNewSession();
  session->gpu = true;

  size_t kernel = 15;
 
  // inputs: filter_height, filter_width, in_channels, out_channels, stride
  Conv2D<float> * conv_layer = new Conv2D<float>(15,15,1,1,2,new GaussianKernelInitializer<float>(15), VALID);
  const char* OUTPUT_BMP_PATH = "./tmp/test/out_gpu.bmp";

  cudaStatus = hipGetLastError();
  checkCudaErrors(cudaStatus);
  
  size_t b_dims[4] = {1, h, w, 1};
  Tensor<float>* bottom = Tensor<float>::CreateTensorGPU(b_dims);
  
  size_t t_dims[4] = {1, h/2-kernel+1, w/2-kernel+1, 1};
  conv_layer->GetTopsDims({b_dims}, {t_dims});
  printf("%d %d %d %d \n", t_dims[0], t_dims[1], t_dims[2], t_dims[3]);
  Tensor<float>* top = Tensor<float>::CreateTensorGPU(t_dims);

  cudaStatus = hipGetLastError();
  checkCudaErrors(cudaStatus);
  
  init_bottom<<<1,1>>>(bottom);
  
  cudaStatus = hipGetLastError();
  checkCudaErrors(cudaStatus);
  
  conv_layer->Forward({bottom}, {top});
  
  cudaStatus = hipGetLastError();
  checkCudaErrors(cudaStatus);
  
  show_top<<<1,1>>>(top);

  cudaStatus = hipGetLastError();
  checkCudaErrors(cudaStatus);
 
  Tensor<float> * top_cpu = Tensor<float>::TensorGPUtoCPU(top);
  cudaStatus = hipGetLastError();
  
  checkCudaErrors(cudaStatus);
  
  bitmap_image img(w/2-kernel+1, h/2-kernel+1);	
  for (int i = 0; i < h/2-kernel+1; i++) {
    for (int j = 0; j < w/2-kernel+1; j++) {
      unsigned val = (unsigned) top_cpu->at(0, i, j, 0);
      img.set_pixel(j, i, val, val, val);
    }
  }
  img.save_image(OUTPUT_BMP_PATH);
  delete conv_layer;
  delete top_cpu;
  hipFree(bottom);
  hipFree(top);
}


int main() {
  test_conv2d_cpu();
  test_conv2d_gpu();
}
