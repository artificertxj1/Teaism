#include "hip/hip_runtime.h"
#include <stdio.h>
#include "basics/tensor.cu"
#include "initializers/gaussian_kernel_initializer.cu"
#include <assert.h>
#include <cmath>
#include <vector>
#include "basics/session.hpp"
#include "layers/conv2d.cu"
#include "utils/utils.cu"
#include "utils/bitmap_image.hpp"



__global__ void init_bottom(Tensor<float> * bottom) {
  for(int b = 0; b < bottom->GetDims()[0]; b++) {
    for(int c = 0; c < bottom->GetDims()[1]; c++) {
      for(int i = 0; i < bottom->GetDims()[2]; i++) {
        for(int j = 0; j < bottom->GetDims()[3]; j++) {
          int b_idx[4] = {b, c, i, j};
          bottom->at(b_idx) = (float) ((i+j+c) % 255);
        }
      }
    }
  }
}

__global__ void show_top(Tensor<float>* top) {
  size_t h = top->GetDims()[1];
  size_t w = top->GetDims()[2];
  for (int i = 0; i < h; i++) {
    for (int j = 0; j < w; j++) {
      printf("%f ", top->at(0, i, j, 0));
    }
    printf("\n");
  }
}


void test_conv2d_cpu() {
  printf("Example code for conv2d cpu\n");
  size_t h = 400;
  size_t w = 400;

  Session* session = Session::GetNewSession();
  session->gpu = false;
 
  // inputs: filter_height, filter_width, in_channels, out_channels, stride
  Conv2D<float> * conv_layer = new Conv2D<float>(15,15,1,1,2,new GaussianKernelInitializer<float>(15));
  const char* OUTPUT_BMP_PATH = "./tmp/test/out.bmp";

  size_t b_dims[4] = {1, h, w, 1};
  Tensor<float>* bottom = Tensor<float>::CreateTensorCPU(b_dims);
  size_t t_dims[4] = {1, h/2, w/2, 1};
  Tensor<float>* top = Tensor<float>::CreateTensorCPU(t_dims);

  for(int i = 0; i < h; i++) {
  	for(int j = 0; j < w; j++) {
  	  int b_idx[4] = {0, i, j, 0};
  	  bottom->at(b_idx) = (float) ((i+j) % 255);
  	}
  }
  conv_layer->Forward({bottom}, {top});

  bitmap_image img(w/2, h/2);
  for (int i = 0; i < h/2; i++) {
    for (int j = 0; j < w/2; j++) {
      unsigned val = (unsigned) top->at(0, i, j, 0);
      img.set_pixel(j, i, val, val, val);
    }
  }
  img.save_image(OUTPUT_BMP_PATH);
  delete conv_layer;
}

/*
void test_conv2d_gpu() {
  printf("Example code for conv2d gpu\n");
  hipError_t cudaStatus = hipSetDevice(0);
  checkCudaErrors(cudaStatus);

  size_t h = 400;
  size_t w = 400;

  Session* session = Session::GetNewSession();
  session->gpu = true;

  size_t kernel = 15;
 
  // inputs: filter_height, filter_width, in_channels, out_channels, stride
  Conv2D<float> * conv_layer = new Conv2D<float>(15,15,1,1,2,new GaussianKernelInitializer<float>(15), VALID);
  const char* OUTPUT_BMP_PATH = "./tmp/test/out_gpu.bmp";

  cudaStatus = hipGetLastError();
  checkCudaErrors(cudaStatus);
  
  size_t b_dims[4] = {1, h, w, 1};
  Tensor<float>* bottom = Tensor<float>::CreateTensorGPU(b_dims);
  
  size_t t_dims[4] = {1, h/2-kernel+1, w/2-kernel+1, 1};
  conv_layer->GetTopsDims({b_dims}, {t_dims});
  printf("%d %d %d %d \n", t_dims[0], t_dims[1], t_dims[2], t_dims[3]);
  Tensor<float>* top = Tensor<float>::CreateTensorGPU(t_dims);

  cudaStatus = hipGetLastError();
  checkCudaErrors(cudaStatus);
  
  init_bottom<<<1,1>>>(bottom);
  
  cudaStatus = hipGetLastError();
  checkCudaErrors(cudaStatus);
  
  conv_layer->Forward({bottom}, {top});
  
  cudaStatus = hipGetLastError();
  checkCudaErrors(cudaStatus);
  
  show_top<<<1,1>>>(top);

  cudaStatus = hipGetLastError();
  checkCudaErrors(cudaStatus);
 
  Tensor<float> * top_cpu = Tensor<float>::TensorGPUtoCPU(top);
  cudaStatus = hipGetLastError();
  
  checkCudaErrors(cudaStatus);
  
  bitmap_image img(w/2-kernel+1, h/2-kernel+1);	
  for (int i = 0; i < h/2-kernel+1; i++) {
    for (int j = 0; j < w/2-kernel+1; j++) {
      unsigned val = (unsigned) top_cpu->at(0, i, j, 0);
      img.set_pixel(j, i, val, val, val);
    }
  }
  img.save_image(OUTPUT_BMP_PATH);
  delete conv_layer;
  delete top_cpu;
  hipFree(bottom);
  hipFree(top);
}*/



void test_conv2d_gpu() {
  printf("Example code for conv2d gpu\n");
  hipError_t cudaStatus = hipSetDevice(0);
  checkCudaErrors(cudaStatus);

  Session* session = Session::GetNewSession();
  session->gpu = true;
  session->batch_size = 64;

  size_t kernel = 15;
  // inputs: filter_height, filter_width, in_channels, out_channels, stride
  Conv2D<float> conv_layer = Conv2D<float>(kernel,kernel,32,64,1, new GaussianKernelInitializer<float>(0.1), SAME);

  size_t b_dims[4] = {session->batch_size, 14, 14, 32};
  Tensor<float>* bottom = Tensor<float>::CreateTensorGPU(b_dims);
  init_bottom<<<1,1>>>(bottom);

  size_t t_dims[4];
  conv_layer.GetTopsDims({b_dims}, {t_dims});
  printf("%d %d %d %d \n", (int)b_dims[0], (int)b_dims[1], (int)b_dims[2], (int)b_dims[3]);
  printf("%d %d %d %d \n", (int)t_dims[0], (int)t_dims[1], (int)t_dims[2], (int)t_dims[3]);
  Tensor<float>* top = Tensor<float>::CreateTensorGPU(t_dims);
  checkCudaErrors(hipGetLastError());

  startTimer();
  conv_layer.Forward({bottom}, {top});
  checkCudaErrors(hipGetLastError());
  printf("conv layer forward: %3.4f ms \n", stopTimer()); 

  show_top<<<1,1>>>(top);
  hipFree(top);
  hipFree(bottom);
  checkCudaErrors(hipGetLastError());
}

int main() {
  // test_conv2d_cpu();
  test_conv2d_gpu();
}
