#include "hip/hip_runtime.h"

#include "layers/softmax.cu"
#include "basics/tensor.cu"
#include <vector>
#include <assert.h>

__global__ void initial_bottom(Tensor<float>* bottom) {
  const size_t* dims = bottom->GetDims();
  printf("(%d, %d)\n", int(dims[0]), int(dims[3]));
  for (int i = 0; i < int(dims[0]); ++i) {
    for (int j = 0; j < int(dims[3]); ++j) {
      bottom->at(i,0,0,j) = (float) i + j;
      printf("(%d, %d): %f\n", i, j, bottom->at(i,0,0,j));
    }
  }
}

__global__ void show_top(Tensor<float>* top) {
  printf("Printing top data\n");
  for (int i = 0; i < int(top->GetDims()[0]); ++i) {
    for (int j = 0; j < int(top->GetDims()[3]); ++j) {
      printf("(%d, %d): %f\n", i, j, top->at(i,0,0,j));
    }
  }
}

void test_softmax_cpu() {
  printf("Begin test softmax layer CPU\n");
  Session* session = Session::GetNewSession();
  session->gpu = false;


  size_t dims[4] = {2, 1, 1, 3};
  std::vector<Tensor<float>*> bottom;
  bottom.push_back(Tensor<float>::CreateTensorCPU(dims));
  std::vector<Tensor<float>*> top;
  top.push_back(Tensor<float>::CreateTensorCPU(dims));

  printf("(%d, %d)\n", bottom[0]->GetDims()[0], bottom[0]->GetDims()[3]);
  for (size_t i = 0; i < dims[0]; ++i) {
    for (size_t j = 0; j < dims[3]; ++j) {
      bottom[0]->at(i,0,0,j) = (float) i + j;
      printf("(%d, %d): %f\n", i, j, bottom[0]->at(i,0,0,j));
    }
  }

  Softmax<float> softmax_layer;
  softmax_layer.Forward(bottom, top);
  
  printf("Printing bottom data\n");
  for (size_t i = 0; i < dims[0]; ++i) {
    for (size_t j = 0; j < dims[3]; ++j) {
      printf("(%d, %d): %f\n", i, j, bottom[0]->at(i,0,0,j));
    }
  }
  printf("Printing top data\n");
  for (size_t i = 0; i < dims[0]; ++i) {
    for (size_t j = 0; j < dims[3]; ++j) {
      printf("(%d, %d): %f\n", i, j, top[0]->at(i,0,0,j));
    }
  }
}


void test_softmax_gpu() {
  printf("Begin test softmax layer GPU\n");
  Session* session = Session::GetNewSession();
  session->gpu = true;

  Softmax<float> softmax_layer;

  hipError_t cudaStatus = hipSetDevice(0);
  checkCudaErrors(cudaStatus);

  size_t dims[4] = {2, 1, 1, 3};
  std::vector<Tensor<float>*> bottom;
  bottom.push_back(Tensor<float>::CreateTensorGPU(dims));
  std::vector<Tensor<float>*> top;
  top.push_back(Tensor<float>::CreateTensorGPU(dims));

  initial_bottom<<<1,1>>>(bottom[0]);

  cudaStatus = hipGetLastError();
  checkCudaErrors(cudaStatus);
  cudaStatus = hipDeviceSynchronize();
  checkCudaErrors(cudaStatus);

  softmax_layer.Forward(bottom, top);
  
  cudaStatus = hipGetLastError();
  checkCudaErrors(cudaStatus);
  cudaStatus = hipDeviceSynchronize();
  checkCudaErrors(cudaStatus);

  show_top<<<1,1>>>(top[0]);

  cudaStatus = hipGetLastError();
  checkCudaErrors(cudaStatus);

  cudaStatus = hipDeviceSynchronize();
  checkCudaErrors(cudaStatus);
}


int main() {
  test_softmax_cpu();
  test_softmax_gpu();
}
