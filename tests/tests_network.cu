#include <stdio.h>
#include <assert.h>
#include "basics/tensor.cu"
#include "basics/session.hpp"
#include "layers/data.cu"
#include "layers/softmax.cu"
#include "layers/cross_entropy_loss.cu"
#include "layers/pooling.cu"
#include "layers/conv2d.cu"
#include "layers/relu.cu"
#include "layers/fc.cu"
#include "utils/bitmap_image.hpp"

#include "hip/hip_runtime.h"
#include ""
#include "utils/hip/hip_runtime_api.h"
#include "utils/utils.cu"
#include "utils/load_model.hpp"
#include "basics/network1.cu"



void test_network_gpu() {
  Network<float> network({});
}


int main() {
  test_network_gpu();
}



