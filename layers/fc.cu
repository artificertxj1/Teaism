#include "hip/hip_runtime.h"
#ifndef FC_LAYER_CUH_
#define FC_LAYER_CUH_

#include "initializers/const_initializer.cu"
#include "basics/layer.hpp"
#include "utils/utils.cu"
// #include <stdio.h>

#define BLOCKDIM 32

namespace FCGPUKernels {
  template <class Dtype> 
  __global__ void ForwardGPUShared(Tensor<Dtype> * bottom, Tensor<Dtype> * top, Tensor<Dtype> * W_, Tensor<Dtype> * b_) {
    size_t n = bottom->GetDims()[0];
    size_t in_channels = bottom->GetDims()[3];
    size_t out_channels = top->GetDims()[3];
    int bi = (blockDim.x * blockIdx.x) + threadIdx.x; // batch idx
    int o = (blockDim.y * blockIdx.y) + threadIdx.y;  // output idi

    extern __shared__ Dtype s[];

    Dtype* in = s;
    Dtype* w = &s[in_channels*BLOCKDIM];

    for(int j = threadIdx.y; j < in_channels; j+= BLOCKDIM) {
      if(bi < n) {
        in[threadIdx.x*in_channels + j] = bottom->at(bi, 0, 0, j);
      }
    }

    for(int j = threadIdx.x; j < in_channels; j+= BLOCKDIM) {
      if(o < out_channels) {
        w[threadIdx.y*in_channels + j] = W_->at(0,0, blockDim.y * blockIdx.y + threadIdx.y, j);
      }
    }
    __syncthreads();

    if (bi < 0 || bi >= n || o < 0 || o >= out_channels) {
      return;
    }

    Dtype sum = 0;
    for(int i = 0; i < in_channels; i++) {
      sum += in[threadIdx.x*in_channels + i] * w[threadIdx.y*in_channels+i];
    }
    sum += b_->at(0,0,0,o);
    top->at(bi,0,0,o) = sum;
  }

  template <class Dtype>
  __global__ void ForwardGPU(Tensor<Dtype> * bottom, Tensor<Dtype> * top, Tensor<Dtype> * W_, Tensor<Dtype> * b_) {
    size_t n = bottom->GetDims()[0];
    size_t in_channels = bottom->GetDims()[3];
    size_t out_channels = top->GetDims()[3];
    int bi = (blockDim.x * blockIdx.x) + threadIdx.x; // batch idx
    int o = (blockDim.y * blockIdx.y) + threadIdx.y;  // output idi
    
    if (bi < 0 || bi >= n || o < 0 || o >= out_channels) {
      return;
    }
    Dtype sum = 0;
    for(int i = 0; i < in_channels; i++) {
      sum += bottom->at(bi, 0, 0, i) * W_->at(0,0,o,i);
      // sum += bottom->at(bi, 0, 0, i) * w[GetIdx(w_dims, 0, o, i)];
    }
    sum += b_->at(0,0,0,o);
    top->at(bi,0,0,o) = sum;
  }
}


template <class Dtype>
class FC: public Layer<Dtype> {
public:

  FC(size_t in_channels, size_t out_channels, Initializer<Dtype>* initializer = NULL):
      in_channels(in_channels), out_channels(out_channels), initializer_(initializer) {
    size_t w_dims[4] = {1, 1, out_channels, in_channels};
    size_t b_dims[4] = {1, 1, 1, out_channels};
    if (Session::GetSession()->gpu) {
      W_ = Tensor<Dtype>::CreateTensorGPU(w_dims);
      b_ = Tensor<Dtype>::CreateTensorGPU(b_dims);
    } else {
      W_ = Tensor<Dtype>::CreateTensorCPU(w_dims);
      b_ = Tensor<Dtype>::CreateTensorCPU(b_dims);
    }
    InitParams();
  }

  ~FC() {
    if (Session::GetSession()->gpu) {
      if (W_!= NULL) {
        hipFree(W_);
        W_ = NULL;
      }
      if (b_ != NULL) {
        hipFree(b_);
        b_ = NULL;
      }
    } else {
      if(W_ != NULL) {
        delete W_;
        W_ = NULL;
      }
      if(b_ != NULL) {
        delete b_;
        b_ = NULL;
      }
    }
  }


  void Forward(const std::vector<Tensor<Dtype>*> &bottoms, const std::vector<Tensor<Dtype>*> &tops) {
    assert(bottoms.size() == 1);
    assert(tops.size() == 1);
    Tensor<Dtype> * bottom = bottoms[0];
    Tensor<Dtype> * top = tops[0];

    if (Session::GetSession()->gpu) {
      size_t batch_size = Session::GetSession()->batch_size;
      dim3 blocksInGrid(batch_size / BLOCKDIM + 1, out_channels / BLOCKDIM + 1);
      dim3 threadsPerBlock(BLOCKDIM, BLOCKDIM);

      if (in_channels*2 < 384) {
        FCGPUKernels::ForwardGPUShared<<<blocksInGrid,threadsPerBlock, 2*in_channels*BLOCKDIM*sizeof(Dtype)>>>(bottom, top, W_, b_);
      } else {
        FCGPUKernels::ForwardGPU<<<blocksInGrid,threadsPerBlock>>>(bottom, top, W_, b_);
      }
    } else {
      for(int b = 0; b < bottom->GetDims()[0]; b++) {
        for(int o = 0; o < out_channels; o++) {
          Dtype sum = 0;
          for(int i = 0; i < in_channels; i++) {
            sum += bottom->at(b, 0, 0, i) * W_->at(0,0,o,i);
          }
          sum += b_->at(0,0,0,o);
          top->at(b,0,0,o) = sum;
        }
      }
    }
  }

  void GetTopsDims(const std::vector<size_t*> &bottoms_dims, 
                   const std::vector<size_t*> &tops_dims) {
    assert(bottoms_dims.size());
    assert(tops_dims.size());
    size_t * b_dims = bottoms_dims[0];
    assert(b_dims[1] == 1);
    assert(b_dims[2] == 1);
    size_t * t_dims = tops_dims[0];
    t_dims[0] = b_dims[0];
    t_dims[1] = 1;
    t_dims[2] = 1;
    t_dims[3] = out_channels;
  }


  const size_t in_channels;
  const size_t out_channels;

  Tensor<Dtype>* W_;
  Tensor<Dtype>* b_;
private:
  const Initializer<Dtype>* initializer_;
  void InitParams() {
    if (initializer_!=NULL) {
      initializer_->Initialize(W_, b_, Session::GetSession()->gpu);
    } else {
      ConstInitializer<Dtype>(1.0, 1.0).Initialize(W_, b_, Session::GetSession()->gpu);
    }
  }
};



#endif  // FC_LAYER_CUH_
