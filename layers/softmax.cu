#include "hip/hip_runtime.h"

#ifndef SOFTMAX_LAYER_CUH_
#define SOFTMAX_LAYER_CUH_

#include <assert.h>
#include <math.h>
#include "basics/layer.hpp"
#include "basics/tensor.cu"
#include "basics/session.hpp"

// TODO: implement CUDA kernel for backward()

#define BLOCKDIM 32

namespace SoftmaxGPUKernels {

  template <class Dtype>
  __global__ void ForwardGPUKernel(Tensor<Dtype>* bottom, Tensor<Dtype>* top) {
    const int batch_idx = threadIdx.x;
    const int batch_size = int(bottom->GetDims()[0]);
    const int nchannels = int(bottom->GetDims()[3]);

    Dtype denominator = 0;
    for (int j = 0; j < nchannels; ++j) {
      top->at(batch_idx,0,0,j) = (Dtype) exp(bottom->at(batch_idx,0,0,j));
      denominator += top->at(batch_idx,0,0,j);
    }
    assert(denominator != 0);
    for (int j = 0; j < nchannels; ++j) {
      top->at(batch_idx,0,0,j) = top->at(batch_idx,0,0,j) / denominator;
    }
  }

  template <class Dtype>
  __global__ void ForwardGPU(Tensor<Dtype>* bottom, Tensor<Dtype>* top) {
    assert(bottom->GetDims()[1] == 1);  // The dimension of the 2nd channel should be 1
    assert(bottom->GetDims()[2] == 1);  // The dimension of the 3rd channel should be 1
    assert(bottom->GetDims()[0] == top->GetDims()[0]);  // bottom channel should be equal to top channel
    assert(bottom->GetDims()[1] == top->GetDims()[1]);
    assert(bottom->GetDims()[2] == top->GetDims()[2]);
    assert(bottom->GetDims()[3] == top->GetDims()[3]);

    SoftmaxGPUKernels::ForwardGPUKernel<Dtype> <<<1,bottom->GetDims()[0]>>>(bottom, top);
  }

}

template <class Dtype>
class Softmax: public Layer<Dtype> {
public:
  Softmax(size_t *dims) {
    if (Session::GetSession()->gpu) {
      top.push_back(Tensor<Dtype>::CreateTensorGPU(dims);
    } else {
      top.push_back(Tensor<Dtype>::CreateTensorCPU(dims);
    }
  }

  ~Softmax() {
    if (Session::GetSession()->gpu) {
      for (auto i: top) {
        hipFree(i);
      }
    } else {
      for (auto i: top) {
        delete i;
      }
    }
}

  void Forward(Tensor<Dtype>* bottom, Tensor<Dtype>* top) {}
  std::vector<Tensor<Dtype>*> Forward(const std::vector<Tensor<Dtype> *> &) {}
  void Forward(const std::vector<Tensor<Dtype>*> &, std::vector<Tensor<Dtype>*> &);

  // void Backward(Tensor& bottom, Tensor& top, Tensor& gradient) {}

private:
  std::vector<Tensor<Dtype>*> top;
};

template <class Dtype>
void Softmax<Dtype>::Forward(const std::vector<Tensor<Dtype>*> &bottom, std::vector<Tensor<Dtype>*> &top) {
  assert(bottom.size() == 1);  // Need only one bottom tensor
  assert(top.size() == 1);  // Need only one bottom tensor

  if (Session::GetSession()->gpu) {
    SoftmaxGPUKernels::ForwardGPU<Dtype><<<1, 1>>>(bottom[0], top[0]); 
  } else {
    assert(bottom[0]->GetDims()[1] == 1);  // The dimension of the 2nd channel should be 1
    assert(bottom[0]->GetDims()[2] == 1);  // The dimension of the 3rd channel should be 1
    assert(bottom[0]->GetDims()[0] == top[0]->GetDims()[0]);  // bottom channel should be equal to top channel
    assert(bottom[0]->GetDims()[1] == top[0]->GetDims()[1]);
    assert(bottom[0]->GetDims()[2] == top[0]->GetDims()[2]);
    assert(bottom[0]->GetDims()[3] == top[0]->GetDims()[3]);

    const size_t batch_size = bottom[0]->GetDims()[0];
    const size_t nchannels = bottom[0]->GetDims()[3];

    Dtype denominator;
    for (int i = 0; i < batch_size; ++i) {
      denominator = 0;
      for (int j = 0; j < nchannels; ++j) {
        top[0]->at(i,0,0,j) = (Dtype) exp(bottom[0]->at(i,0,0,j));
        denominator += top[0]->at(i,0,0,j);
      }
      for (int j = 0; j < nchannels; ++j) {
        top[0]->at(i,0,0,j) = top[0]->at(i,0,0,j) / denominator;
      }
    }
  }
}


#endif  // SOFTMAX_LAYER_CUH_
