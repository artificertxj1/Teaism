#include "hip/hip_runtime.h"

#ifndef CONV2D_LAYER_CUH_
#define CONV2D_LAYER_CUH_

#include <assert.h>
#include <stdio.h>
#include "basics/layer.hpp"
#include "basics/tensor.cu"
#include "basics/session.hpp"
#include "basics/initializer.hpp"
#include "hip/hip_runtime.h"
#include ""
#include "utils/hip/hip_runtime_api.h"
#include "utils/helper_string.h"
#include "initializers/gaussian_kernel_initializer.cu"

// TODO: implement CUDA kernel for backward()

#define BLOCKDIM 32

enum PADDING {SAME, VALID};

namespace ConvGPUKernels {

  template <class Dtype>
  __global__ void ForwardGPUKernel(Tensor<Dtype> * bottom, Tensor<Dtype> * top, Tensor<Dtype> * W, Tensor<Dtype> * b, int bi, int o, int stride, PADDING padding) {
    // bi is the index of the tensor
    // o is the output channel
    size_t kernel_height;
    size_t kernel_width;
    kernel_height = W->GetDims()[0];
    kernel_width = W->GetDims()[1];
    


    int x_top = (blockDim.x * blockIdx.x) + threadIdx.x;
    int y_top = (blockDim.y * blockIdx.y) + threadIdx.y;
    int x = x_top*stride;
    int y = y_top*stride;

    if (!bottom->isValidIdx(bi, o, y, x) || !top->isValidIdx(bi, o, y_top, x_top)) {
      return;
    }

    if (padding==VALID) {
      x = kernel_width/2 + x_top*stride;
      y = kernel_height/2 + y_top*stride;
      if (!bottom->isValidIdx(bi, o, y, x) || !top->isValidIdx(bi, o, y_top, x_top) || !bottom->isValidIdx(bi, o, y + kernel_height/2, x + kernel_height/2)) {
        return;
      }
    }

    int idx[4] = {bi, o, y, x};
    size_t in_channels = bottom->GetDims()[1];
    Dtype sum = 0.0;
    for(int c = 0; c < in_channels; c++) {
      for(int i = 0; i < kernel_height; i++) {
        for(int j = 0; j < kernel_width; j++) {
          // (n, hei, wid, channel),   // (hei, wid, input, output)
          sum += bottom->atPadding(bi, c, y+i-int(kernel_height/2), x+j-int(kernel_width/2)) * W->at(i, j, c, o);
        }
      }
    }
    sum += b->at(0, 0, 0, o);
    top->at(bi, o, y_top, x_top) = sum;
  }

  template <class Dtype>
  __global__ void ForwardGPU2(Tensor<Dtype> * bottom, Tensor<Dtype> * top, Tensor<Dtype> * W_, Tensor<Dtype> * b_, int stride, PADDING padding=SAME) {
    size_t n = bottom->GetDims()[0];
    size_t out_channels = top->GetDims()[1];
    size_t hei = top->GetDims()[2];
    size_t wid = top->GetDims()[3];

    int b = (blockDim.x * blockIdx.x) + threadIdx.x;
    int o = (blockDim.y * blockIdx.y) + threadIdx.y;
    
    if(b < 0 || b >= n || o < 0 || o >= out_channels) return;

    dim3 blocksInGrid(wid / BLOCKDIM + 1, hei / BLOCKDIM + 1);
    dim3 threadsPerBlock(BLOCKDIM, BLOCKDIM);
    
    ConvGPUKernels::ForwardGPUKernel<Dtype><<<blocksInGrid, threadsPerBlock>>>(bottom, top, W_, b_, b, o, stride, padding);
  }  



  template <class Dtype>
  __global__ void ForwardGPU(Tensor<Dtype> * bottom, Tensor<Dtype> * top, Tensor<Dtype> * W_, Tensor<Dtype> * b_, int stride, PADDING padding=SAME) {
    size_t n = bottom->GetDims()[0];
    size_t out_channels = top->GetDims()[1];
    size_t hei = top->GetDims()[2];
    size_t wid = top->GetDims()[3];
  
    dim3 blocksInGrid(n / BLOCKDIM + 1, out_channels / BLOCKDIM + 1);
    dim3 threadsPerBlock(BLOCKDIM, BLOCKDIM);
    
    ConvGPUKernels::ForwardGPU2<Dtype><<<blocksInGrid, threadsPerBlock>>>(bottom, top, W_, b_, stride, padding);

    // dim3 blocksInGrid(wid / BLOCKDIM + 1, hei / BLOCKDIM + 1);
    // dim3 threadsPerBlock(BLOCKDIM, BLOCKDIM);
    // for (int b = 0; b < n; b++) {
    //   for (int o = 0; o < out_channels; o++) {
    //     ConvGPUKernels::ForwardGPUKernel<Dtype><<<blocksInGrid, threadsPerBlock>>>(bottom, top, W_, b_, b, o, stride, padding);
    //   }
    // }
  }
}

template <class Dtype>
class Conv2D: public Layer<Dtype> {
public:
  // use the same initializer to initialize W_ and b_
  Conv2D(size_t kernel_height, size_t kernel_width, size_t in_channels, 
    size_t out_channels, size_t stride, Initializer<Dtype>* initializer = NULL, PADDING _padding=SAME);

  ~Conv2D();

  void Forward(const std::vector<Tensor<Dtype>*> &bottoms, const std::vector<Tensor<Dtype>*> &tops);

  void GetTopsDims(const std::vector<size_t*> &bottoms_dims, 
                  const std::vector<size_t*> &tops_dims);

  const size_t kernel_height;
  const size_t kernel_width;
  const size_t in_channels;
  const size_t out_channels;
  const size_t stride;
  const PADDING padding;
private:
  Tensor<Dtype>* W_;
  Tensor<Dtype>* b_;
  const Initializer<Dtype>* initializer_;
  void InitParams(); 
};


template<class Dtype> 
Conv2D<Dtype>::Conv2D(size_t kernel_height, size_t kernel_width, size_t in_channels, 
    size_t out_channels, size_t stride, Initializer<Dtype>* initializer, PADDING _padding):
      kernel_height(kernel_height), kernel_width(kernel_width),
      in_channels(in_channels), out_channels(out_channels), 
      stride(stride), initializer_(initializer),
      padding(_padding) {
  size_t w_dims[4] = {kernel_height, kernel_width, in_channels, out_channels};
  size_t b_dims[4] = {1, 1, 1, out_channels};
  if (Session::GetSession()->gpu) {
    W_ = Tensor<Dtype>::CreateTensorGPU(w_dims);
    b_ = Tensor<Dtype>::CreateTensorGPU(b_dims);
  } else {
    W_ = Tensor<Dtype>::CreateTensorCPU(w_dims);
    b_ = Tensor<Dtype>::CreateTensorCPU(b_dims);
  }
  InitParams();
}


template<class Dtype>
Conv2D<Dtype>::~Conv2D() {
  if (Session::GetSession()->gpu) {
    if (W_!= NULL) {
      hipFree(W_);
      W_ = NULL;
    }
    if (b_ != NULL) {
      hipFree(b_);
      b_ = NULL;
    }
  } else {
    if(W_ != NULL) {
      delete W_;
      W_ = NULL;
    }
    if(b_ != NULL) {
      delete b_;
      b_ = NULL;
    }
  }
}

template<class Dtype>
void Conv2D<Dtype>::Forward(const std::vector<Tensor<Dtype>*> &bottoms, const std::vector<Tensor<Dtype>*> &tops) {
  assert(bottoms.size()==1);
  assert(tops.size()==1);
  Tensor<Dtype> * bottom = bottoms[0];
  Tensor<Dtype> * top = tops[0];

  if (Session::GetSession()->gpu) {
    ConvGPUKernels::ForwardGPU<<<1,Session::GetSession()->batch_size>>>(bottom, top, W_, b_, stride, padding);
  } else {
    size_t b_hei = bottom->GetDims()[2];
    size_t b_wid = bottom->GetDims()[3];

    for(int b = 0; b < bottom->GetDims()[0]; b++) {
      for(int o = 0; o < out_channels; o++) {
        if(padding==SAME) {
          for(int y = 0, y_top = 0; y < b_hei; y += stride, y_top += 1) {
            for(int x = 0, x_top = 0; x < b_wid; x += stride, x_top += 1) {
              // batch idx b, output layer o, pixel (x, y)
              // top->at({b, o, y, x}) = 
              Dtype sum = 0.0;
              for(int c = 0; c < in_channels; c++) {
                for(int i = 0; i < kernel_height; i++) {
                  for(int j = 0; j < kernel_width; j++) {
                    // (n, hei, wid, channel),   // (hei, wid, input, output)
                    int b_idx[4] = {b, c, y+i-int(kernel_height/2), x+j-int(kernel_width/2)};
                    int t_idx[4] = {i, j, c, o};
                    sum += bottom->atPadding(b_idx) * W_->at(t_idx);
                  }
                }
              }
              sum += b_->at(0,0,0,o);
              top->at(b, o, y_top, x_top) = sum;
            }
          }
        } else if (padding==VALID) {
          for(int y = kernel_height/2, y_top = 0; y < b_hei - kernel_height/2; y += stride, y_top += 1) {
            for(int x = kernel_width/2, x_top = 0; x < b_wid - kernel_width/2; x += stride, x_top += 1) {
              // batch idx b, output layer o, pixel (x, y)
              // top->at({b, o, y, x}) = 
              int idx[4] = {b, o, y, x};
              Dtype sum = 0.0;
              for(int c = 0; c < in_channels; c++) {
                for(int i = 0; i < kernel_height; i++) {
                  for(int j = 0; j < kernel_width; j++) {
                    // (n, channel, hei, wid),   // (hei, wid, input, output)
                    sum += bottom->atPadding(b, c,  y +i-int(kernel_height/2), x+j-int(kernel_width/2)) * W_->at(i, j, c, o);
                  }
                }
              }
              int b_idx[4] = {0,0,0,o};
              sum += b_->at(b_idx);
              int t_idx[4] = {b, o, y_top, x_top};        
              top->at(t_idx) = sum;
            }
          }
        }
      }
    }
  }
}



template<class Dtype>
void Conv2D<Dtype>::GetTopsDims(const std::vector<size_t*> &bottoms_dims, 
                      const std::vector<size_t*> &tops_dims) {
  assert(bottoms_dims.size());
  assert(tops_dims.size());
  size_t * b_dims = bottoms_dims[0];
  size_t * t_dims = tops_dims[0];
  if(padding == SAME) {
    t_dims[0] = b_dims[0];
    t_dims[1] = out_channels;
    t_dims[2] = b_dims[2]/stride;
    t_dims[3] = b_dims[3]/stride;
    printf("%d %d\n",b_dims[2]/stride, b_dims[3]/stride);
  } else if(padding == VALID) {
    t_dims[0] = b_dims[0];
    t_dims[1] = out_channels;
    t_dims[2] = b_dims[2]/stride - kernel_height + 1;
    t_dims[3] = b_dims[3]/stride - kernel_width + 1;
  }
}


template<class Dtype>
void Conv2D<Dtype>::InitParams() {
  if (initializer_!=NULL) {
    initializer_->Initialize(W_, b_, Session::GetSession()->gpu);
  } else {
    GaussianKernelInitializer<Dtype>((Dtype)(kernel_width+kernel_height)/2).Initialize(W_, b_, Session::GetSession()->gpu);
  }
}


#endif  // CONV2D_LAYER_CUH_
