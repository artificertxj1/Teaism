#include "hip/hip_runtime.h"
#ifndef RELU_LAYER_CUH_
#define RELU_LAYER_CUH_

#include "basics/layer.hpp"
#include "basics/tensor.cu"
#include "basics/session.hpp"

#define BLOCKDIM 32

template <class Dtype>
__global__ void relu(Tensor<Dtype> * bottom, Tensor<Dtype> * top, int bi, int o) {
  // bi is the index of the tensor
  // o is the output channel
  int x_top = (blockDim.x * blockIdx.x) + threadIdx.x;
  int y_top = (blockDim.y * blockIdx.y) + threadIdx.y;
  int x = x_top;
  int y = y_top;
  if (!bottom->isValidIdx(bi, y, x, o) || !top->isValidIdx(bi, y_top, x_top, o)) {
    return;
  }
  Dtype val = bottom->at(bi, y, x, o);
  top->at(bi, y_top, x_top, o) = (val >= 0 ? val : 0);
}

template <class Dtype>
__global__ void ReluForwardGPU(Tensor<Dtype> * bottom, Tensor<Dtype> * top) {
  size_t n = bottom->GetDims()[0];
  size_t hei = top->GetDims()[1];
  size_t wid = top->GetDims()[2];
  size_t out_channels = top->GetDims()[3];

  dim3 blocksInGrid(wid / BLOCKDIM + 1, hei / BLOCKDIM + 1);
  dim3 threadsPerBlock(BLOCKDIM, BLOCKDIM);
  for (int b = 0; b < n; b++) {
    for (int o = 0; o < out_channels; o++) {
      relu<Dtype><<<blocksInGrid, threadsPerBlock>>>(bottom, top, b, o);
    }
  }
}


template <class Dtype>
class Relu: public Layer<Dtype> {
public:
  void Forward(Tensor<Dtype> * bottom, Tensor<Dtype> * top) {
    if (Session::GetSession()->gpu) {
      ReluForwardGPU<<<1,1>>>(bottom, top);
    } else {
      for(int b = 0; b < bottom->GetDims()[0]; b++) {
        for(int o = 0; o < bottom->GetDims()[3]; o++) {
          for(int x = 0; x < bottom->GetDims()[2]; x += 1) {
            for(int y = 0; y < bottom->GetDims()[1]; y += 1) {
              Dtype val = bottom->at(b, y, x, o);
              top->at(b, y, x, o) = (val >= 0 ? val : 0);
            }
          }
        }
      }
    }
  }
};

#endif 