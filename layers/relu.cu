#include "hip/hip_runtime.h"
#ifndef POOLING_LAYER_CUH_
#define POOLING_LAYER_CUH_

template <class Dtype>
__global__ void relu(Tensor<Dtype> * bottom, Tensor<Dtype> * top, int bi, int o, size_t size, POOLING_TYPE type) {
  // bi is the index of the tensor
  // o is the output channel
  int x_top = (blockDim.x * blockIdx.x) + threadIdx.x;
  int y_top = (blockDim.y * blockIdx.y) + threadIdx.y;
  int x = x_top;
  int y = y_top;
  if (!bottom->isValidIdx(bi, y, x, o) || !top->isValidIdx(bi, y_top, x_top, o)) {
    return;
  }
  Dtype val = bottom->at(b, y, x, o);
  top->at(b, y_top, x_top, o) = (val >= 0 ? val : 0);
}

template <class Dtype>
__global__ void ReluForwardGPU(Tensor<Dtype> * bottom, Tensor<Dtype> * top) {
  dim3 blocksInGrid(wid / BLOCKDIM + 1, hei / BLOCKDIM + 1);
  dim3 threadsPerBlock(BLOCKDIM, BLOCKDIM);
  for (int b = 0; b < n; b++) {
    for (int o = 0; o < out_channels; o++) {
      pool<Dtype><<<blocksInGrid, threadsPerBlock>>>(bottom, top, b, o);
    }
  }
}


template <class Dtype>
class Relu: public Layer<Dtype> {
public:
  void Forward(Tensor<Dtype> * bottom, Tensor<Dtype> * top) {
    if (Session::GetSession()->gpu) {
      ReluForwardGPU<<<1,1>>>(bottom, top);
    } else {
      for(int b = 0; b < bottom->GetDims()[0]; b++) {
        for(int o = 0; o < bottom->GetDims()[3]; o++) {
          for(int x = 0, x_top = 0; x < bottom->GetDims()[2]; x += 1) {
            for(int y = 0, y_top = 0; y < bottom->GetDims()[1]; y += 1) {
              Dtype val = bottom->at(b, y, x, o);
              top->at(b, y, x, o) = (val >= 0 ? val : 0);
            }
          }
        }
      }
    }
  }
};

#define 