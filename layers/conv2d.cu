#include "hip/hip_runtime.h"

#ifndef CONV2D_LAYER_CUH_
#define CONV2D_LAYER_CUH_

#include <assert.h>
#include <stdio.h>
#include "basics/layer.hpp"
#include "basics/tensor.cu"
#include "basics/session.hpp"
#include "basics/initializer.hpp"
#include "hip/hip_runtime.h"
#include ""
#include "utils/hip/hip_runtime_api.h"
#include "utils/helper_string.h"
#include "initializers/gaussian_kernel_initializer.cu"

// TODO: implement CUDA kernel for backward()

#define BLOCKDIM 32

enum PADDING {SAME, VALID};

namespace ConvGPUKernels {

  template <class Dtype>
  __global__ void ForwardGPUKernel(Tensor<Dtype> * bottom, Tensor<Dtype> * top, Tensor<Dtype> * W, Tensor<Dtype> * b, int bi, int o, int stride, PADDING padding) {
    // bi is the index of the tensor
    // o is the output channel
    int x_top = (blockDim.x * blockIdx.x) + threadIdx.x;
    int y_top = (blockDim.y * blockIdx.y) + threadIdx.y;
    size_t kernel_height = W->GetDims()[0];
    size_t kernel_width = W->GetDims()[1];
    int x = x_top*stride;
    int y = y_top*stride;

    if (!bottom->isValidIdx(bi, y, x, o) || !top->isValidIdx(bi, y_top, x_top, o)) {
      return;
    }

    if (padding==VALID) {
      x = kernel_width/2 + x_top*stride;
      y = kernel_height/2 + y_top*stride;
      if (!bottom->isValidIdx(bi, y, x, o) || !top->isValidIdx(bi, y_top, x_top, o) || !bottom->isValidIdx(bi, y + kernel_height/2, x + kernel_height/2, o)) {
        return;
      }
    }

    int idx[4] = {bi, y, x, o};
    size_t in_channels = bottom->GetDims()[3];
    Dtype sum = 0.0;
    for(int c = 0; c < in_channels; c++) {
      for(int i = 0; i < kernel_height; i++) {
        for(int j = 0; j < kernel_width; j++) {
          // (n, hei, wid, channel),   // (hei, wid, input, output)
          sum += bottom->atPadding(idx[0], idx[1]+i-int(kernel_height/2), idx[2]+j-int(kernel_width/2), c) * W->at(i, j, c, idx[3]);
        }
      }
    }
    sum += b->at(0, 0, 0, 0);
    top->at(bi, y_top, x_top, o) = sum;
  }

  template <class Dtype>
  __global__ void ForwardGPU(Tensor<Dtype> * bottom, Tensor<Dtype> * top, Tensor<Dtype> * W_, Tensor<Dtype> * b_, int stride, PADDING padding=SAME) {
    size_t n = bottom->GetDims()[0];
    size_t hei = top->GetDims()[1];
    size_t wid = top->GetDims()[2];
    size_t out_channels = top->GetDims()[3];

    dim3 blocksInGrid(wid / BLOCKDIM + 1, hei / BLOCKDIM + 1);
    dim3 threadsPerBlock(BLOCKDIM, BLOCKDIM);
    for (int b = 0; b < n; b++) {
      for (int o = 0; o < out_channels; o++) {
        ConvGPUKernels::ForwardGPUKernel<Dtype><<<blocksInGrid, threadsPerBlock>>>(bottom, top, W_, b_, b, o, stride, padding);
      }
    }
  }
}

template <class Dtype>
class Conv2D: public Layer<Dtype> {
public:
  // use the same initializer to initialize W_ and b_
  Conv2D(size_t kernel_height, size_t kernel_width, size_t in_channels, 
    size_t out_channels, size_t stride, Initializer<Dtype>* initializer = NULL, PADDING _padding=SAME):
      kernel_height(kernel_height), kernel_width(kernel_width),
      in_channels(in_channels), out_channels(out_channels), 
      stride(stride), initializer_(initializer),
      padding(_padding) {
    size_t w_dims[4] = {kernel_height, kernel_width, in_channels, out_channels};
    size_t b_dims[4] = {1, 1, 1, out_channels};  
    if (Session::GetSession()->gpu) {
      W_ = Tensor<Dtype>::CreateTensorGPU(w_dims);
      b_ = Tensor<Dtype>::CreateTensorGPU(b_dims);
    } else {
      W_ = Tensor<Dtype>::CreateTensorCPU(w_dims);
      b_ = Tensor<Dtype>::CreateTensorCPU(b_dims);
    }
    InitParams();
  }

  ~Conv2D() {
    if (Session::GetSession()->gpu) {
      if (W_!= NULL) {
        hipFree(W_);
        W_ = NULL;
      }
      if (b_ != NULL) {
        hipFree(b_);
        b_ = NULL;
      }
    } else {
      if(W_ != NULL) {
        delete W_;
        W_ = NULL;
      }
      if(b_ != NULL) {
        delete b_;
        b_ = NULL;
      }
    }
  }

  virtual void Forward(const std::vector<Tensor<Dtype>*> &bottoms, const std::vector<Tensor<Dtype>*> &tops) {
    assert(bottoms.size()==1);
    assert(tops.size()==1);
    Tensor<Dtype> * bottom = bottoms[0];
    Tensor<Dtype> * top = tops[0];

    if (Session::GetSession()->gpu) {
      ConvGPUKernels::ForwardGPU<<<1,1>>>(bottom, top, W_, b_, stride, padding);
    } else {
      for(int b = 0; b < bottom->GetDims()[0]; b++) {
        for(int o = 0; o < out_channels; o++) {
          if(padding==SAME) {
            for(int x = 0, x_top = 0; x < bottom->GetDims()[2]; x += stride, x_top += 1) {
              for(int y = 0, y_top = 0; y < bottom->GetDims()[1]; y += stride, y_top += 1) {
                // batch idx b, output layer o, pixel (x, y)
                // top->at({b, y, x, o}) = 
                int idx[4] = {b, y, x, o};
                Dtype sum = 0.0;
                for(int c = 0; c < in_channels; c++) {
                  for(int i = 0; i < kernel_height; i++) {
                    for(int j = 0; j < kernel_width; j++) {
                      // (n, hei, wid, channel),   // (hei, wid, input, output)
                      int b_idx[4] = {idx[0], idx[1]+i-int(kernel_height/2), idx[2]+j-int(kernel_width/2), c};
                      int t_idx[4] = {i, j, c, idx[3]};
                      sum += bottom->atPadding(b_idx) * W_->at(t_idx);
                    }
                  }
                }
                int b_idx[4] = {0,0,0,0};
                sum += b_->at(b_idx);
                int t_idx[4] = {b, y_top, x_top, o};
                
                top->at(t_idx) = sum;
              }
            }
          } else if (padding==VALID) {
            for(int x = kernel_width/2, x_top = 0; x < bottom->GetDims()[2] - kernel_width/2; x += stride, x_top += 1) {
              for(int y = kernel_height/2, y_top = 0; y < bottom->GetDims()[1] - kernel_height/2; y += stride, y_top += 1) {
                // batch idx b, output layer o, pixel (x, y)
                // top->at({b, y, x, o}) = 
                int idx[4] = {b, y, x, o};
                Dtype sum = 0.0;
                for(int c = 0; c < in_channels; c++) {
                  for(int i = 0; i < kernel_height; i++) {
                    for(int j = 0; j < kernel_width; j++) {
                      // (n, hei, wid, channel),   // (hei, wid, input, output)
                      int b_idx[4] = {idx[0], idx[1]+i-int(kernel_height/2), idx[2]+j-int(kernel_width/2), c};
                      int t_idx[4] = {i, j, c, idx[3]};
                      sum += bottom->atPadding(b_idx) * W_->at(t_idx);
                    }
                  }
                }
                int b_idx[4] = {0,0,0,0};
                sum += b_->at(b_idx);
                int t_idx[4] = {b, y_top, x_top, o};        
                top->at(t_idx) = sum;
              }
            }
          }
        }
      }
    }
  }

  // void Backward(Tensor& bottom, Tensor& top, Tensor& gradient) {}

  const size_t kernel_height;
  const size_t kernel_width;
  const size_t in_channels;
  const size_t out_channels;
  const size_t stride;
  const PADDING padding;
private:
  Tensor<Dtype>* W_;
  Tensor<Dtype>* b_;
  const Initializer<Dtype>* initializer_;
  void InitParams() {
    if (initializer_!=NULL) {
      initializer_->Initialize(W_, b_, Session::GetSession()->gpu);
    } else {
      GaussianKernelInitializer<Dtype>((Dtype)(kernel_width+kernel_height)/2).Initialize(W_, b_, Session::GetSession()->gpu);
    }
  }
};


#endif  // CONV2D_LAYER_CUH_
