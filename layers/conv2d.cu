#include "hip/hip_runtime.h"

#ifndef CONV2D_LAYER_CUH_
#define CONV2D_LAYER_CUH_

#include <assert.h>
#include <stdio.h>
#include "basics/layer.hpp"
#include "basics/tensor.cu"
#include "basics/session.hpp"
#include "basics/initializer.hpp"
#include "hip/hip_runtime.h"
#include ""
#include "utils/hip/hip_runtime_api.h"
#include "utils/helper_string.h"
#include "initializers/gaussian_kernel_initializer.cu"
#include "utils/utils.cu"

// TODO: implement CUDA kernel for backward()

#define BLOCKDIM 32

enum PADDING {SAME, VALID};

namespace ConvGPUKernels {
  

  template <class Dtype>
  __global__ void ForwardGPUKernel2(Tensor<Dtype> * bottom, Tensor<Dtype> * top, Tensor<Dtype> * W, Tensor<Dtype> * b, int hs, int ws, int stride, PADDING padding) {
    // find bi & o
    int bi = blockIdx.y/hs;
    int o = blockIdx.x/ws;

    int hi = (blockIdx.y % hs);
    // int hi = blockIdx.y;
    int wi = (blockIdx.x % ws);

    int y_top = hi * blockDim.y + threadIdx.y;
    int x_top = wi * blockDim.x + threadIdx.x;

    int x = x_top * stride;
    int y = y_top * stride;

    size_t in_channels = bottom->GetDims()[3];

    size_t kernel_height = W->GetDims()[0];
    size_t kernel_width = W->GetDims()[1];

    if (!bottom->isValidIdx(bi, y, x, o) || !top->isValidIdx(bi, y_top, x_top, o)) {
      return;
    }

    extern __shared__ Dtype s[];
    Dtype * k = s;
    const size_t * w_dims = W->GetDims();
    if(threadIdx.x < kernel_width && threadIdx.y < kernel_height) {
      for(int c = 0; c < in_channels; c++) {
        k[GetIdx(w_dims, threadIdx.y, threadIdx.x, c)] = W->at(threadIdx.y, threadIdx.x, c, o);
      }
    }
    __syncthreads();


    if (padding==VALID) {
      x = kernel_width/2 + x_top*stride;
      y = kernel_height/2 + y_top*stride;
      if (!bottom->isValidIdx(bi, y, x, o) || !top->isValidIdx(bi, y_top, x_top, o) || !bottom->isValidIdx(bi, y + kernel_height/2, x + kernel_height/2, o)) {
        return;
      }
    }

    Dtype sum = 0.0;
    for(int i = 0; i < kernel_height; i++) {
      for(int j = 0; j < kernel_width; j++) {
        for(int c = 0; c < in_channels; c++) {
          // (n, hei, wid, channel),   // (hei, wid, input, output)
          // sum += bottom->atPadding(bi, y+i-int(kernel_height/2), x+j-int(kernel_width/2), c) * W->at(i, j, c, o);
          sum += bottom->atPadding(bi, y+i-int(kernel_height/2), x+j-int(kernel_width/2), c) * k[GetIdx(w_dims, i, j, c)];
        }
      }
    }	
    sum += b->at(0, 0, 0, o);
    top->at(bi, y_top, x_top, o) = sum;
  }


  template <class Dtype>
  __global__ void ForwardGPUKernel(Tensor<Dtype> * bottom, Tensor<Dtype> * top, Tensor<Dtype> * W, Tensor<Dtype> * b, int bi, int oi, int stride, PADDING padding) {
    // bi is the index of the tensor
    // o is the output channel
    int x_top = (blockDim.x * blockIdx.x) + threadIdx.x;
    int y_top = (blockDim.y * blockIdx.y) + threadIdx.y;
    // int o = (blockDim.z * blockIdx.z) + threadIdx.z;
    int o = oi;

    size_t kernel_height = W->GetDims()[0];
    size_t kernel_width = W->GetDims()[1];
    int x = x_top*stride;
    int y = y_top*stride;
    
    size_t in_channels = bottom->GetDims()[3];   
    // size_t bs = bottom->GetDims()[0];

    // if(bi < 0 || bi >= bs) return;

    if (!bottom->isValidIdx(bi, y, x, o) || !top->isValidIdx(bi, y_top, x_top, o)) {
      return;
    }
    extern __shared__ Dtype s[];
    Dtype * k = s;
    const size_t * w_dims = W->GetDims();
    if(threadIdx.x < kernel_width && threadIdx.y < kernel_height) {
      for(int c = 0; c < in_channels; c++) {
        k[GetIdx(w_dims, threadIdx.y, threadIdx.x, c)] = W->at(threadIdx.y, threadIdx.x, c, o);
      }
    }
    __syncthreads();


    if (padding==VALID) {
      x = kernel_width/2 + x_top*stride;
      y = kernel_height/2 + y_top*stride;
      if (!bottom->isValidIdx(bi, y, x, o) || !top->isValidIdx(bi, y_top, x_top, o) || !bottom->isValidIdx(bi, y + kernel_height/2, x + kernel_height/2, o)) {
        return;
      }
    }

    Dtype sum = 0.0;
    for(int i = 0; i < kernel_height; i++) {
      for(int j = 0; j < kernel_width; j++) {
        for(int c = 0; c < in_channels; c++) {
          // (n, hei, wid, channel),   // (hei, wid, input, output)
          // sum += bottom->atPadding(bi, y+i-int(kernel_height/2), x+j-int(kernel_width/2), c) * W->at(i, j, c, o);
          sum += bottom->atPadding(bi, y+i-int(kernel_height/2), x+j-int(kernel_width/2), c) * k[GetIdx(w_dims, i, j, c)];
        }
      }
    }
    sum += b->at(0, 0, 0, o);
    top->at(bi, y_top, x_top, o) = sum;
  }
}

template <class Dtype>
class Conv2D: public Layer<Dtype> {
public:
  // use the same initializer to initialize W_ and b_
  Conv2D(size_t kernel_height, size_t kernel_width, size_t in_channels, 
    size_t out_channels, size_t stride, Initializer<Dtype>* initializer = NULL, PADDING _padding=SAME);

  ~Conv2D();

  void Forward(const std::vector<Tensor<Dtype>*> &bottoms, const std::vector<Tensor<Dtype>*> &tops);

  void GetTopsDims(const std::vector<size_t*> &bottoms_dims, 
                  const std::vector<size_t*> &tops_dims);

  const size_t kernel_height;
  const size_t kernel_width;
  const size_t in_channels;
  const size_t out_channels;
  const size_t stride;
  const PADDING padding;
private:
  Tensor<Dtype>* W_;
  Tensor<Dtype>* b_;
  const Initializer<Dtype>* initializer_;
  void InitParams(); 
};



template<class Dtype> 
Conv2D<Dtype>::Conv2D(size_t kernel_height, size_t kernel_width, size_t in_channels, 
    size_t out_channels, size_t stride, Initializer<Dtype>* initializer, PADDING _padding):
      kernel_height(kernel_height), kernel_width(kernel_width),
      in_channels(in_channels), out_channels(out_channels), 
      stride(stride), initializer_(initializer),
      padding(_padding) {
  size_t w_dims[4] = {kernel_height, kernel_width, in_channels, out_channels};
  size_t b_dims[4] = {1, 1, 1, out_channels};
  if (Session::GetSession()->gpu) {
    W_ = Tensor<Dtype>::CreateTensorGPU(w_dims);
    b_ = Tensor<Dtype>::CreateTensorGPU(b_dims);
  } else {
    W_ = Tensor<Dtype>::CreateTensorCPU(w_dims);
    b_ = Tensor<Dtype>::CreateTensorCPU(b_dims);
  }
  InitParams();
}


template<class Dtype>
Conv2D<Dtype>::~Conv2D() {
  if (Session::GetSession()->gpu) {
    if (W_!= NULL) {
      hipFree(W_);
      W_ = NULL;
    }
    if (b_ != NULL) {
      hipFree(b_);
      b_ = NULL;
    }
  } else {
    if(W_ != NULL) {
      delete W_;
      W_ = NULL;
    }
    if(b_ != NULL) {
      delete b_;
      b_ = NULL;
    }
  }
}

template<class Dtype>
void Conv2D<Dtype>::Forward(const std::vector<Tensor<Dtype>*> &bottoms, const std::vector<Tensor<Dtype>*> &tops) {
  assert(bottoms.size()==1);
  assert(tops.size()==1);
  Tensor<Dtype> * bottom = bottoms[0];
  Tensor<Dtype> * top = tops[0];

  if (Session::GetSession()->gpu) {
    size_t t_dims[4];
    Tensor<float>::GetTensorGPUDims(top, t_dims);
    size_t bs = Session::GetSession()->batch_size;
    size_t hei = t_dims[1];
    size_t wid = t_dims[2];

    size_t hs = hei/BLOCKDIM + 1;
    size_t ws = wid/BLOCKDIM + 1;

    dim3 blocksInGrid(ws*out_channels, hs*bs);
    dim3 threadsPerBlock(BLOCKDIM, BLOCKDIM);
    // for(int b = 0; b < bs; b++) {
    ConvGPUKernels::ForwardGPUKernel2<Dtype><<<blocksInGrid, threadsPerBlock, kernel_height*kernel_width*in_channels*sizeof(Dtype)>>>(bottom, top, W_, b_, hs, ws, stride, padding);
    // }
/*
    size_t t_dims[4];
    Tensor<float>::GetTensorGPUDims(top, t_dims);
    size_t bs = Session::GetSession()->batch_size;
    size_t hei = t_dims[1];
    size_t wid = t_dims[2];

    size_t hs = hei/BLOCKDIM + 1;
    size_t ws = wid/BLOCKDIM + 1;

    dim3 blocksInGrid(ws*out_channels, hs*bs);
    dim3 threadsPerBlock(BLOCKDIM, BLOCKDIM);

    ConvGPUKernels::ForwardGPUKernel2<Dtype><<<blocksInGrid, threadsPerBlock, kernel_height*kernel_width*in_channels*sizeof(Dtype)>>>(bottom, top, W_, b_, hs, ws, stride, padding);
*/




    // old
  	/*
    size_t t_dims[4];
    Tensor<float>::GetTensorGPUDims(top, t_dims);
    size_t bs = Session::GetSession()->batch_size;
    size_t hei = t_dims[1];
    size_t wid = t_dims[2];
    // dim3 blocksInGrid(wid / BLOCKDIM + 1, hei / BLOCKDIM * 4 + 1, out_channels/4 + 1);
    // dim3 threadsPerBlock(BLOCKDIM, BLOCKDIM/4, 4);
    dim3 blocksInGrid(wid / BLOCKDIM + 1, hei / BLOCKDIM + 1);
    dim3 threadsPerBlock(BLOCKDIM, BLOCKDIM);
    // dims3 blocksInGrid(wid*hei*out_channels*bs/BLOCKDIM/BLOCKDIM+1);
    // dims3 threadsPerBlock(BLOCKDIM*BLOCKDIM);
    for (int b = 0; b < bs; b++) {
      for (int o = 0; o < out_channels; o++) {
//        ConvGPUKernels::ForwardGPUKernel<Dtype><<<blocksInGrid, threadsPerBlock, kernel_height*kernel_width*in_channels*sizeof(Dtype)+(BLOCKDIM+kernel_height)*(BLOCKDIM+kernel_width)*in_channels*sizeof(Dtype)>>>(bottom, top, W_, b_, b, o, stride, padding);
        ConvGPUKernels::ForwardGPUKernel<Dtype><<<blocksInGrid, threadsPerBlock, kernel_height*kernel_width*in_channels*sizeof(Dtype)>>>(bottom, top, W_, b_, b, o, stride, padding);
        // ConvGPUKernels::ForwardGPUKernel2<Dtype><<<blocksInGrid, threadsPerBlock, kernel_height*kernel_width*in_channels*sizeof(Dtype)>>>(bottom, top, W_, b_, 0, 0, stride, padding);
      }
    }*/
  } else {
    for(int b = 0; b < bottom->GetDims()[0]; b++) {
      for(int o = 0; o < out_channels; o++) {
        if(padding==SAME) {
          for(int x = 0, x_top = 0; x < bottom->GetDims()[2]; x += stride, x_top += 1) {
            for(int y = 0, y_top = 0; y < bottom->GetDims()[1]; y += stride, y_top += 1) {
              // batch idx b, output layer o, pixel (x, y)
              // top->at({b, y, x, o}) = 
              int idx[4] = {b, y, x, o};
              Dtype sum = 0.0;
              for(int c = 0; c < in_channels; c++) {
                for(int i = 0; i < kernel_height; i++) {
                  for(int j = 0; j < kernel_width; j++) {
                    // (n, hei, wid, channel),   // (hei, wid, input, output)
                    int b_idx[4] = {idx[0], idx[1]+i-int(kernel_height/2), idx[2]+j-int(kernel_width/2), c};
                    int t_idx[4] = {i, j, c, idx[3]};
                    sum += bottom->atPadding(b_idx) * W_->at(t_idx);
                  }
                }
              }
              int b_idx[4] = {0,0,0,o};
              sum += b_->at(b_idx);
              int t_idx[4] = {b, y_top, x_top, o};
              
              top->at(t_idx) = sum;
            }
          }
        } else if (padding==VALID) {
          for(int x = kernel_width/2, x_top = 0; x < bottom->GetDims()[2] - kernel_width/2; x += stride, x_top += 1) {
            for(int y = kernel_height/2, y_top = 0; y < bottom->GetDims()[1] - kernel_height/2; y += stride, y_top += 1) {
              // batch idx b, output layer o, pixel (x, y)
              // top->at({b, y, x, o}) = 
              int idx[4] = {b, y, x, o};
              Dtype sum = 0.0;
              for(int c = 0; c < in_channels; c++) {
                for(int i = 0; i < kernel_height; i++) {
                  for(int j = 0; j < kernel_width; j++) {
                    // (n, hei, wid, channel),   // (hei, wid, input, output)
                    int b_idx[4] = {idx[0], idx[1]+i-int(kernel_height/2), idx[2]+j-int(kernel_width/2), c};
                    int t_idx[4] = {i, j, c, idx[3]};
                    sum += bottom->atPadding(b_idx) * W_->at(t_idx);
                  }
                }
              }
              int b_idx[4] = {0,0,0,o};
              sum += b_->at(b_idx);
              int t_idx[4] = {b, y_top, x_top, o};        
              top->at(t_idx) = sum;
            }
          }
        }
      }
    }
  }
}



template<class Dtype>
void Conv2D<Dtype>::GetTopsDims(const std::vector<size_t*> &bottoms_dims, 
                      const std::vector<size_t*> &tops_dims) {
  assert(bottoms_dims.size());
  assert(tops_dims.size());
  size_t * b_dims = bottoms_dims[0];
  size_t * t_dims = tops_dims[0];
  if(padding == SAME) {
    t_dims[0] = b_dims[0];
    t_dims[1] = b_dims[1]/stride;
    t_dims[2] = b_dims[2]/stride;
    t_dims[3] = out_channels;
  } else if(padding == VALID) {
    t_dims[0] = b_dims[0];
    t_dims[1] = b_dims[1]/stride - kernel_height + 1;
    t_dims[2] = b_dims[2]/stride - kernel_width + 1;
    t_dims[3] = out_channels;
  }
}


template<class Dtype>
void Conv2D<Dtype>::InitParams() {
  if (initializer_!=NULL) {
    initializer_->Initialize(W_, b_, Session::GetSession()->gpu);
  } else {
    GaussianKernelInitializer<Dtype>((Dtype)(kernel_width+kernel_height)/2).Initialize(W_, b_, Session::GetSession()->gpu);
  }
}


#endif  // CONV2D_LAYER_CUH_
