#include "hip/hip_runtime.h"
#ifndef TENSOR_CUH_
#define TENSOR_CUH_

#include <assert.h>
#include <cstdlib>
#include <numeric>
#include <functional>
#include "basics/session.hpp"
#include "hip/hip_runtime.h"
#include "utils/hip/hip_runtime_api.h"
#include "stdio.h"

/* 
4D Tensor
*/
template<class Dtype>
class Tensor {

public:

  __host__ __device__ ~Tensor() {
    if(data_array_ != NULL) {
      delete [] data_array_;
    }
  }

  __host__ __device__ unsigned GetIdx(const int* idx) const {
    unsigned out_idx = 0;
    for (int i = 0; i < 4; i++)
      out_idx = out_idx*dims_[i] + idx[i];
    return out_idx;
  }

  __host__ __device__ const size_t* GetDims() const {
    return dims_;
  }

  __host__ __device__ Dtype* GetDataPtr() const {
    return data_array_;
  }

  __host__ __device__ void SetDataPtr(Dtype* data_array_ptr) {
    data_array_ = data_array_ptr;
  }

  __host__ __device__ Dtype& at(const int i0, const int i1, const int i2, const int i3) {
    int idx[4] = {i0, i1, i2, i3};
    return at(idx);
  }

  __host__ __device__ Dtype& at(const int* idx) {
    assert(isValidIdx(idx));
    return data_array_[GetIdx(idx)];
  }

  __host__ __device__ const Dtype atPadding(const int i0, const int i1, const int i2, const int i3) {
    int idx[4] = {i0, i1, i2, i3};
    return atPadding(idx);
  }

  __host__ __device__ const Dtype atPadding(int* idx, Dtype default_val = 0.0) const {
    if (!isValidIdx(idx)) return default_val;
    return data_array_[GetIdx(idx)];
  }

  __host__ __device__ bool isValidIdx(const int i0, const int i1, const int i2, const int i3) {
    int idx[4] = {i0, i1, i2, i3};
    return isValidIdx(idx);
  }

  __host__ __device__ bool isValidIdx(const int* idx) const {
    for(int i = 0; i < 4; i++) {
      // printf("%d\n", idx[i]);
      if(idx[i] < 0 || idx[i] >= dims_[i]) return false;
    }
    return true;
  }

  __host__ __device__ size_t size() const {
    return len_;
  }

  __host__ __device__ void AllocateDataArray() {
    if(data_array_ == NULL) {
      data_array_ = new Dtype[len_];
    }
  }


  // host functions
  __host__ Tensor<Dtype> * GetGPUPtr() const {
    if(gpu_ptr_ == NULL) {
      hipMalloc((void**)&gpu_ptr_, sizeof(Tensor<Dtype>));
      hipMemcpy(gpu_ptr_, this, sizeof(Tensor<Dtype>), hipMemcpyHostToDevice);
    }
    return gpu_ptr_;
  }
  __host__ void AllocateDataArrayGPU();

  __host__ static Tensor<Dtype>* CreateTensorGPU(size_t* dims, bool allocate_memory = true) {
    Tensor<Dtype> tensor_cpu(dims);
    Tensor<Dtype>* tensor_gpu;
    hipMalloc((void**)&tensor_gpu, sizeof(Tensor<Dtype>));
    hipMemcpy(tensor_gpu, &tensor_cpu, sizeof(Tensor<Dtype>), hipMemcpyHostToDevice);
    if (allocate_memory) {
      AllocateDataArrayGPU(tensor_gpu);
    }
    return tensor_gpu;
  }

  __host__ static Tensor<Dtype>* CreateTensorCPU(size_t* dims, bool allocate_memory = true) {
    Tensor<Dtype> * tensor_cpu = new Tensor(dims);
    if (allocate_memory) {
      tensor_cpu->AllocateDataArray();
    }
    return tensor_cpu;
  }

  __host__ static Tensor<Dtype> * TensorGPUtoCPU(Tensor<Dtype> * tensor_gpu) {
    Tensor<Dtype> * tensor_cpu = (Tensor<Dtype> *)malloc(sizeof(Tensor<Dtype>));
    hipMemcpy(tensor_cpu, tensor_gpu, sizeof(Tensor<Dtype>), hipMemcpyDeviceToHost);
    Dtype * data_array_ = (Dtype*) malloc(tensor_cpu->size()*sizeof(Dtype));
    hipMemcpy(data_array_, tensor_cpu->data_array_, tensor_cpu->size() * sizeof(Dtype), hipMemcpyDeviceToHost);
    tensor_cpu->SetDataPtr(data_array_);
    return tensor_cpu;
  }

  __host__ static void AllocateDataArrayGPU(Tensor<Dtype> * tensor_gpu);

  Dtype* data_array_;
//private:
  __host__ __device__ Tensor(size_t dims[4]): gpu_ptr_(NULL), data_array_(NULL) {
    len_ = dims[0] * dims[1] * dims[2] * dims[3];
    dims_[0] = dims[0];
    dims_[1] = dims[1];
    dims_[2] = dims[2];
    dims_[3] = dims[3];
  }
  __host__ __device__ Tensor(): gpu_ptr_(NULL), data_array_(NULL) {}

  size_t dims_[4];
  size_t len_;
  
  Tensor<Dtype> * gpu_ptr_;
};

/*
template<class Dtype>
__global__ void allocate_tensor_dataarray(Tensor<Dtype> * tensor_gpu) {
  tensor_gpu->AllocateDataArray();
}

template<class Dtype>
__host__ void Tensor<Dtype>::AllocateDataArrayGPU() {
  Tensor<Dtype>* _gpu_ptr = GetGPUPtr();
  allocate_tensor_dataarray<Dtype><<<1, 1>>>(_gpu_ptr);
}*/

template<class Dtype>
__host__ void Tensor<Dtype>::AllocateDataArrayGPU(Tensor<Dtype> * tensor_gpu) {
    size_t * len = (size_t *) malloc(sizeof(size_t));
    hipMemcpy(len, &tensor_gpu->len_, sizeof(size_t), hipMemcpyDeviceToHost);
    Dtype* data_array_gpu;
    hipMalloc((void**)&(data_array_gpu), sizeof(Dtype)*(*len));
    hipMemcpy(&(tensor_gpu->data_array_), &data_array_gpu, sizeof(Dtype*), hipMemcpyHostToDevice);
}



#endif // TENSOR_CUH_
