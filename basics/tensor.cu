#ifndef TENSOR_CUH_
#define TENSOR_CUH_

#include <assert.h>
#include <cstdlib>
#include <numeric>
#include <functional>
#include "basics/session.hpp"
#include "hip/hip_runtime.h"
#include "utils/hip/hip_runtime_api.h"
#include "stdio.h"

/* 
4D Tensor
*/
template<class Dtype>
class Tensor {

public:

  __host__ __device__ unsigned GetIdx(const int* idx) const {
    unsigned out_idx = 0;
    for (int i = 0; i < 4; i++)
      out_idx = out_idx*dims_[i] + idx[i];
    return out_idx;
  }

  __host__ __device__ const size_t* GetDims() const {
    return dims_;
  }

  __host__ __device__ Dtype* GetDataPtr() const {
    return data_array_;
  }

  __host__ __device__ void SetDataPtr(Dtype* data_array_ptr) {
    data_array_ = data_array_ptr;
  }

  __host__ __device__ Dtype& at(const int i0, const int i1, const int i2, const int i3) {
    int idx[4] = {i0, i1, i2, i3};
    return at(idx);
  }

  __host__ __device__ Dtype& at(const int* idx) {
    assert(isValidIdx(idx));
    return data_array_[GetIdx(idx)];
  }

  __host__ __device__ const Dtype atPadding(const int i0, const int i1, const int i2, const int i3) {
    int idx[4] = {i0, i1, i2, i3};
    return atPadding(idx);
  }

  __host__ __device__ const Dtype atPadding(int* idx, Dtype default_val = 0.0) const {
    if (!isValidIdx(idx)) return default_val;
    return data_array_[GetIdx(idx)];
  }

  __host__ __device__ bool isValidIdx(const int i0, const int i1, const int i2, const int i3) {
    int idx[4] = {i0, i1, i2, i3};
    return isValidIdx(idx);
  }

  __host__ __device__ bool isValidIdx(const int* idx) const {
    for(int i = 0; i < 4; i++) {
      if(idx[i] < 0 || idx[i] >= dims_[i]) return false;
    }
    return true;
  }

  __host__ __device__ size_t size() const {
    return len_;
  }

  __host__ static Tensor<Dtype>* CreateTensorGPU(size_t* dims, bool allocate_memory = true);
  __host__ static Tensor<Dtype>* CreateTensorCPU(size_t* dims, bool allocate_memory = true);
  __host__ static Tensor<Dtype> * TensorGPUtoCPU(Tensor<Dtype> * tensor_gpu);
  __host__ static Tensor<Dtype> * TensorCPUtoGPU(Tensor<Dtype> * tensor_cpu);
  __host__ static void AllocateDataArrayGPU(Tensor<Dtype> * tensor_gpu);
  __host__ static void AllocateDataArrayCPU(Tensor<Dtype> * tensor_cpu);

  __host__ __device__ ~Tensor() {
    if(data_array_ != NULL) {
      delete [] data_array_;
    }
  }
  

private:
  __host__ __device__ Tensor(size_t dims[4]): data_array_(NULL) {
    len_ = dims[0] * dims[1] * dims[2] * dims[3];
    dims_[0] = dims[0];
    dims_[1] = dims[1];
    dims_[2] = dims[2];
    dims_[3] = dims[3];
  }

  Dtype* data_array_;
  size_t dims_[4];
  size_t len_;
};


// Create CPU/GPU Tensor
template<class Dtype>
__host__ Tensor<Dtype>* Tensor<Dtype>::CreateTensorCPU(size_t* dims, bool allocate_memory) {
  Tensor<Dtype> * tensor_cpu = new Tensor(dims);
  if (allocate_memory) {
    AllocateDataArrayCPU(tensor_cpu);
  }
  return tensor_cpu;
}

template<class Dtype>
__host__ Tensor<Dtype>* Tensor<Dtype>::CreateTensorGPU(size_t* dims, bool allocate_memory) {
  Tensor<Dtype> tensor_cpu(dims);
  Tensor<Dtype>* tensor_gpu;
  hipMalloc((void**)&tensor_gpu, sizeof(Tensor<Dtype>));
  hipMemcpy(tensor_gpu, &tensor_cpu, sizeof(Tensor<Dtype>), hipMemcpyHostToDevice);

  if (allocate_memory) {
    AllocateDataArrayGPU(tensor_gpu);
  }
  return tensor_gpu;
}

template<class Dtype>
__host__ Tensor<Dtype> * Tensor<Dtype>::TensorGPUtoCPU(Tensor<Dtype> * tensor_gpu) {
  Tensor<Dtype> * tensor_cpu = (Tensor<Dtype> *)malloc(sizeof(Tensor<Dtype>));
  hipMemcpy(tensor_cpu, tensor_gpu, sizeof(Tensor<Dtype>), hipMemcpyDeviceToHost);
  Dtype * data_array_ = (Dtype*) malloc(tensor_cpu->size()*sizeof(Dtype));
  hipMemcpy(data_array_, tensor_cpu->GetDataPtr(), tensor_cpu->size() * sizeof(Dtype), hipMemcpyDeviceToHost);
  tensor_cpu->SetDataPtr(data_array_);
  return tensor_cpu;
}

template<class Dtype>
__host__ Tensor<Dtype> * Tensor<Dtype>::TensorCPUtoGPU(Tensor<Dtype> * tensor_cpu) {
  Tensor<Dtype> * tensor_gpu; 
  hipMalloc((void **)&tensor_gpu, sizeof(Tensor<Dtype>));
  hipMemcpy(tensor_gpu, tensor_cpu, sizeof(Tensor<Dtype>), hipMemcpyHostToDevice);

  Dtype* data_array;
  hipMalloc((void**) &data_array, sizeof(Dtype)*tensor_cpu->size());
  hipMemcpy(data_array, tensor_cpu->GetDataPtr(), sizeof(Dtype)*tensor_cpu->size(), hipMemcpyHostToDevice);
  hipMemcpy(&tensor_gpu->data_array_, &data_array, sizeof(Dtype*), hipMemcpyHostToDevice);

  return tensor_gpu;
}

// Allocate Memory 
template<class Dtype>
__host__ void Tensor<Dtype>::AllocateDataArrayGPU(Tensor<Dtype> * tensor_gpu) {
    size_t * len = (size_t *) malloc(sizeof(size_t));
    hipMemcpy(len, &tensor_gpu->len_, sizeof(size_t), hipMemcpyDeviceToHost);
    Dtype* data_array_gpu;
    hipMalloc((void**)&(data_array_gpu), sizeof(Dtype)*(*len));
    hipMemcpy(&(tensor_gpu->data_array_), &data_array_gpu, sizeof(Dtype*), hipMemcpyHostToDevice);
}

template<class Dtype>
__host__ void Tensor<Dtype>::AllocateDataArrayCPU(Tensor<Dtype> * tensor_cpu) {
  if (tensor_cpu->data_array_ == NULL) {
    tensor_cpu->data_array_ = new Dtype[tensor_cpu->len_];
  }
}


#endif // TENSOR_CUH_
