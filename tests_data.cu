#include "hip/hip_runtime.h"

#include "layers/data.cu"
#include <vector>
#include <assert.h>

__global__ void show_top(Tensor<float>* top) {
  printf("%f\n", top->at(0,1020,531,0));
  printf("%f\n", top->at(1,1020,531,0));
  printf("%f\n", top->at(0,1001,555,1));
  printf("%f\n", top->at(1,1001,555,1));
  printf("%f\n", top->at(0,1000,500,2));
  printf("%f\n", top->at(1,1000,500,2));
}

void test_data_cpu() {
  printf("Begin test data layer CPU\n");
  Session* session = Session::GetNewSession();
  session->gpu = false;

  Data<float> data_layer(2, "/home/jyh/github/Teaism/tmp/test/img_list.txt");

  printf("Frist image: %s\n", data_layer.img_list[0].c_str());
  printf("First label: %d\n", data_layer.lab_list[0]);
  printf("Number of data: %d\n", data_layer.lab_list.size());
  printf("Image width: %d\n", data_layer.img_w);
  printf("Image height: %d\n", data_layer.img_h);
  assert(data_layer.img_list.size() == data_layer.lab_list.size());

  std::vector<Tensor<float>* > top;
  top = data_layer.Forward();
  
  printf("%f\n", top[0]->at(0,1020,531,0));
  printf("%f\n", top[0]->at(1,1020,531,0));
  printf("%f\n", top[0]->at(0,1001,555,1));
  printf("%f\n", top[0]->at(1,1001,555,1));
  printf("%f\n", top[0]->at(0,1000,500,2));
  printf("%f\n", top[0]->at(1,1000,500,2));
}

void test_data_gpu() {
  printf("Begin test data layer GPU\n");
  Session* session = Session::GetNewSession();
  session->gpu = true;

  Data<float> data_layer(2, "/home/jyh/github/Teaism/tmp/test/img_list.txt");

  printf("Frist image: %s\n", data_layer.img_list[0].c_str());
  printf("First label: %d\n", data_layer.lab_list[0]);
  printf("Number of data: %d\n", data_layer.lab_list.size());
  printf("Image width: %d\n", data_layer.img_w);
  printf("Image height: %d\n", data_layer.img_h);
  assert(data_layer.img_list.size() == data_layer.lab_list.size());

  hipError_t cudaStatus = hipSetDevice(0);
  checkCudaErrors(cudaStatus);

  std::vector<Tensor<float>* > top;
  top = data_layer.Forward();
  
  cudaStatus = hipGetLastError();
  checkCudaErrors(cudaStatus);

  show_top<<<1,1>>>(top[0]);

  cudaStatus = hipGetLastError();
  checkCudaErrors(cudaStatus);

  cudaStatus = hipDeviceSynchronize();
  checkCudaErrors(cudaStatus);
}


int main() {
  test_data_cpu();
  test_data_gpu();
}
